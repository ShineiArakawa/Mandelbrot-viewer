#include "hip/hip_runtime.h"
#include <Mandelbrot/kernel.cuh>

__global__ void kernel(
    uchar4* deviceArray,
    const double minX,
    const double minY,
    const double maxX,
    const double maxY,
    const int width,
    const int height,
    const int maxIter,
    const double threshold,
    const bool isEnabledSmoothing,
    const double alphaCoeff,
    const bool isEnabledSinuidalColor,
    const double density) {
  const int indexX = threadIdx.x + blockIdx.x * blockDim.x;
  const int indexY = threadIdx.y + blockIdx.y * blockDim.y;
  const int offset = indexX + indexY * width;

  if (indexX < width && indexY < height) {
    const double dX = (maxX - minX) / (double)(width - 1);
    const double dY = (maxY - minY) / (double)(height - 1);
    const double coordX = (double)indexX * dX + minX;
    const double coordY = (double)indexY * dY + minY;

    double zReal = 0.0;
    double zImag = 0.0;
    int stoppedIter = 0;
    double radius = 0.0;

    for (int iter = 0; iter < maxIter; iter++) {
      radius = sqrt(zReal * zReal + zImag * zImag);

      if (radius > threshold) {
        stoppedIter = iter;
        break;
      }

      const double tmp_zReal = zReal * zReal - zImag * zImag + coordX;
      const double tmp_zImag = 2.0 * zReal * zImag + coordY;
      zReal = tmp_zReal;
      zImag = tmp_zImag;
    }

    int R, G, B, A = 0;
    double alpha = 0.0;

    if (isEnabledSmoothing) {
      const double log2Val = log(2.0);
      const double nu = log(log(radius) / log2Val) / log2Val;
      alpha = alphaCoeff * ((double)stoppedIter + 1.0 - nu);
    } else {
      alpha = alphaCoeff * (double)stoppedIter;
    }

    if (isEnabledSinuidalColor) {
      alpha = alpha * density;
      alpha = log(alpha + 1.0);

      const double factorR = (cos((alpha * 2.0 - 1.0) * M_PI) + 1.0) * 0.5;
      const double factorG = (cos((alpha * 2.0 - 0.75) * M_PI) + 1.0) * 0.5;
      const double factorB = (cos((alpha * 2.0 - 0.5) * M_PI) + 1.0) * 0.5;

      R = (int)(factorR * 255.0);
      G = (int)(factorG * 255.0);
      B = (int)(factorB * 255.0);
      A = 255;
    } else {
      const int pixelValue = max(min((int)(alpha * 255.0), 255), 0);
      R = pixelValue;
      G = pixelValue;
      B = pixelValue;
      A = 255;
    }

    deviceArray[offset].x = R;
    deviceArray[offset].y = G;
    deviceArray[offset].z = B;
    deviceArray[offset].w = A;
  }
}

void launchCUDAKernel(
    uchar4* deviceArray,
    const double minX,
    const double minY,
    const double maxX,
    const double maxY,
    const int width,
    const int height,
    const int maxIter,
    const double threshold,
    const bool isEnabledSmoothing,
    const double alphaCoeff,
    const bool isEnabledSinuidalColor,
    const double density) {
  dim3 blockDim(16, 16);
  dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

  kernel<<<gridDim, blockDim>>>(
      deviceArray,
      minX,
      minY,
      maxX,
      maxY,
      width,
      height,
      maxIter,
      threshold,
      isEnabledSmoothing,
      alphaCoeff,
      isEnabledSinuidalColor,
      density);
}
